#include "hip/hip_runtime.h"
/**********************************************************************
 *  IzerRaster - CUDA backend                                         *
 *  - Rasterizare triunghiuri cu Z-buffer                              *
 *  - OPTIONAL: sampling textură RGBA8                                 *
 *********************************************************************/
#include "render.h"

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cfloat>      // FLT_MAX
#include <cmath>       // sqrt, floor
#include <algorithm>   // min
#include <cstdint>  

/* ====================================================================
   -- Buffere de frame                                                 */
static uint32_t* d_colorBuffer = nullptr;   // ARGB8   [width * height]
static float*    d_depthBuffer = nullptr;   // float32 [width * height]
static int       frameWidth  = 0;
static int       frameHeight = 0;

/* ==== Config pentru lansare kernel ================================= */
static dim3 threadsPerBlock(16, 16);
static dim3 blocksPerGrid (1 , 1 );

// __device__ Light d_light;
// __device__ glm::vec3 d_cameraPos;
// __device__ Material d_material;

/* ====================================================================
   -- Textură globală (read-only)                                      */
__device__ uint32_t* d_texture = nullptr;   // pixel array RGBA8
__device__ int       d_texW    = 0;
__device__ int       d_texH    = 0;
__device__ bool      d_useTex  = false;     // switch sampling ON/OFF

extern "C" void uploadTexture(const uint32_t* devPixels, int w, int h)
{
    hipMemcpyToSymbol(HIP_SYMBOL(d_texture), &devPixels, sizeof(uint32_t*));
    hipMemcpyToSymbol(HIP_SYMBOL(d_texW),    &w,         sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_texH),    &h,         sizeof(int));
}

// extern "C" void uploadLighting(const Light& light, const glm::vec3& camPos, const Material& material) 
// {
//     hipMemcpyToSymbol(HIP_SYMBOL(d_light), &light, sizeof(Light));
//     hipMemcpyToSymbol(HIP_SYMBOL(d_cameraPos), &camPos, sizeof(glm::vec3));
//     hipMemcpyToSymbol(HIP_SYMBOL(d_material), &material, sizeof(Material));
// }

extern "C" void setTexturing(bool enable)
{
    hipMemcpyToSymbol(HIP_SYMBOL(d_useTex), &enable, sizeof(bool));
}

/* ====================================================================
   Device helpers                                                      */
__device__ __forceinline__
uint32_t sampleTexture(float u, float v)
{
    /* clamp / wrap la [0,1] */
    u -= floorf(u);          // wrap (repeat)
    v -= floorf(v);

    int x = int(u * d_texW);
    int y = int((1.f - v) * d_texH);
    x = ::max(0, ::min(x, d_texW  - 1));   // ::max = built-in device func
    y = ::max(0, ::min(y, d_texH  - 1));
    return d_texture[y * d_texW + x];
}

/* ====================================================================
   Kernel: clear color + depth                                         */
__global__ static void clearBuffers(uint32_t* colorBuf, float* depthBuf,
                                    int width, int height,
                                    uint32_t clearColor, float clearDepth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    int idx = y * width + x;
    colorBuf[idx] = clearColor;
    depthBuf[idx] = clearDepth;
}

/* ====================================================================
   Kernel: rasterize triunghi cu Z-buffer + opţional textură           */
__global__ static void rasterizeTri(uint32_t* colorBuf, float* depthBuf,
                                    int width, int height,
                                    float x0, float y0,
                                    float x1, float y1,
                                    float x2, float y2,
                                    float z0, float z1, float z2,
                                    float u0, float v0,
                                    float u1, float v1,
                                    float u2, float v2,
                                    uint32_t flatColor)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    int idx = y * width + x;

    /* --- barycentric ------------------------------------------------ */
    float denom = (y1 - y2) * (x0 - x2) + (x2 - x1) * (y0 - y2);
    if (denom == 0.f) return;                         // degenerate

    float w0 = ((y1 - y2) * (x - x2) + (x2 - x1) * (y - y2)) / denom;
    float w1 = ((y2 - y0) * (x - x2) + (x0 - x2) * (y - y2)) / denom;
    float w2 = 1.f - w0 - w1;

    if (w0 < 0.f || w1 < 0.f || w2 < 0.f) return;     // pixel în afara

    /* --- depth ------------------------------------------------------ */
    float z = w0 * z0 + w1 * z1 + w2 * z2;
    if (z >= depthBuf[idx]) return;                   // test Z

    /* --- culoare ---------------------------------------------------- */

    depthBuf[idx] = z;

    // glm::vec3 p0 = glm::vec3(x0, y0, z0);
    // glm::vec3 p1 = glm::vec3(x1, y1, z1);
    // glm::vec3 p2 = glm::vec3(x2, y2, z2);

    // glm::vec3 fragPos = w0 * p0 + w1 * p1 + w2 * p2;

    // glm::vec3 edge1 = p1 - p0;
    // glm::vec3 edge2 = p2 - p0;

    // glm::vec3 normal = glm::normalize(glm::cross(edge1, edge2));

    // glm::vec3 lightDirection = glm::normalize(d_light.position - fragPos);
    // float diff = fmaxf(glm::dot(normal, lightDirection), 0.0f);

    // glm::vec3 viewDirection = glm::normalize(d_cameraPos - fragPos);
    // glm::vec3 reflectDirection = glm::reflect(-lightDirection, normal);
    // float spec = powf(fmaxf(glm::dot(viewDirection, reflectDirection), 0.0f), d_material.shininess);

    // glm::vec3 diffuse = d_material.diffuseColour * diff * d_light.intensity;
    // glm::vec3 specular = d_material.specularColour * spec * d_light.intensity;
    // glm::vec3 colour = (diffuse + specular) * d_light.colour;

    // colour += 0.5f * d_material.diffuseColour;
    // colour = glm::clamp(colour, 0.0f, 1.0f);

    uint32_t texColor = flatColor;
    if (d_useTex && d_texture) 
    {
        float u = w0 * u0 + w1 * u1 + w2 * u2;
        float v = w0 * v0 + w1 * v1 + w2 * v2;
        texColor = sampleTexture(u, v);
    }

    // float texR = ((texColor >> 16) & 0xFF) / 255.0f;
    // float texG = ((texColor >> 8) & 0xFF) / 255.0f;
    // float texB = (texColor & 0xFF) / 255.0f;

    // float outR = texR * colour.x;
    // float outG = texG * colour.y;
    // float outB = texB * colour.z;

    // uint32_t outColour = (255 << 24) |
    //                      (uint8_t(outR * 255.0f) << 16) |
    //                      (uint8_t(outG * 255.0f) << 8) |
    //                      (uint8_t(outB * 255.0f));
    colorBuf[idx] = texColor;
}

/* ====================================================================
   Init CUDA – alocă buffere                                            */
extern "C" bool initCuda(int width, int height)
{
    frameWidth  = width;
    frameHeight = height;

    size_t colorBytes = size_t(width) * height * sizeof(uint32_t);
    size_t depthBytes = size_t(width) * height * sizeof(float);

    hipError_t ec = hipMalloc(&d_colorBuffer, colorBytes);
    hipError_t ed = hipMalloc(&d_depthBuffer, depthBytes);
    if (ec != hipSuccess || ed != hipSuccess)
    {
        std::fprintf(stderr, "hipMalloc failed: %s\n",
                     hipGetErrorString(ec != hipSuccess ? ec : ed));
        return false;
    }

    /* calcul blocuri */
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int maxT = prop.maxThreadsPerBlock;
    int maxDim = std::min(32, int(std::floor(std::sqrt(float(maxT)))));
    int side = 1; while (side * 2 <= maxDim) side *= 2;

    threadsPerBlock = dim3(side, side);
    blocksPerGrid   = dim3( (width  + side - 1) / side,
                            (height + side - 1) / side );

    std::printf("[CUDA] init OK – \"%s\", block %dx%d, grid %dx%d\n",
                prop.name, side, side, blocksPerGrid.x, blocksPerGrid.y);
    return true;
}

/* ====================================================================
   Render frame: clear + loop triunghiuri                               */
extern "C" void renderFrame(const CudaTri* tris, int numTris,
                            uint32_t* hostPix, float* hostDepth)
{
    /* --- clear ------------------------------------------------------ */
    const uint32_t clr = 0xFF000000u;
    clearBuffers<<<blocksPerGrid, threadsPerBlock>>>(d_colorBuffer,
                                                     d_depthBuffer,
                                                     frameWidth,
                                                     frameHeight,
                                                     clr, FLT_MAX);

    /* --- rasterizare ------------------------------------------------ */
    const uint32_t white = 0xFFFFFFFFu;
    for (int i = 0; i < numTris; ++i)
    {
        const CudaTri& t = tris[i];
        rasterizeTri<<<blocksPerGrid, threadsPerBlock>>>(
            d_colorBuffer, d_depthBuffer,
            frameWidth, frameHeight,
            t.x0, t.y0,  t.x1, t.y1,  t.x2, t.y2,
            t.z0, t.z1,  t.z2,
            t.u0, t.v0,  t.u1, t.v1,  t.u2, t.v2,
            white);
    }

    /* --- copy back -------------------------------------------------- */
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess)
        std::fprintf(stderr, "Kernel error: %s\n", hipGetErrorString(err));

    size_t colorBytes = size_t(frameWidth) * frameHeight * sizeof(uint32_t);
    size_t depthBytes = size_t(frameWidth) * frameHeight * sizeof(float);

    hipMemcpy(hostPix,   d_colorBuffer, colorBytes, hipMemcpyDeviceToHost);
    hipMemcpy(hostDepth, d_depthBuffer, depthBytes, hipMemcpyDeviceToHost);
}

/* ====================================================================
   Cleanup                                                             */
extern "C" void cleanupCuda()
{
    if (d_colorBuffer) { hipFree(d_colorBuffer); d_colorBuffer = nullptr; }
    if (d_depthBuffer) { hipFree(d_depthBuffer); d_depthBuffer = nullptr; }
    setTexturing(false); 
}
